/*
#define _SIZE_T_DEFINED
#ifndef __CUDACC__
#define __CUDACC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif
*/

#define THREADS_PER_BLOCK 64
#define _SIZE_T_DEFINED
#ifndef __cplusplus
#define __cplusplus
#endif
#ifndef __CUDACC__  
    #define __CUDACC__
#endif

#include <hip/hip_runtime.h>
#include <math_functions.h>

//#include <texture_fetch_functions.h>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <hip/device_functions.h>

//#include <cuda_runtime_api.h>
extern "C"  
{
	// Scale v2, Several blocks with 64 (LUDBLOCK_SCALE_BLOCKSIZE) threads
	__global__ void lud_block_scale_v2(float *a, int M, int k, int end)
	{
		extern __shared__ float ac[];

		int aWidth = M;
		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		ac[threadIdx.x] = a[k * aWidth + k + threadIdx.x]; // Load k row to shared memory, as it is used across threads

		// Sync threads to make sure all other also have loaded values
		__syncthreads();

		//for(int i = k+1 + tx; i < M; i+=blockDim.x) { // Foreach row
		int i = k+1 + tid;
		if (i < M)
		{
			 // Compute scale factor Rik, 1 operation=divide
			float rik = (a[i * aWidth + k] /= ac[0]); // CGMA: 0.5

			for (int c = 1; c < end-k; c++)  // Foreach column value in row
				a[i * aWidth + k + c] -= rik * ac[c]; // CGMA: 1.0
		}
	}
	// Tri.solve v1, Several blocks with 64 (LUDBLOCK_TRIANGULAR_BLOCKSIZE) threads
	__global__ void lud_block_triangular_solve(float *a, int M, int k, int LU_BlockDim)
	{
		extern __shared__ float y[];

		int tx = threadIdx.x;
		int tid = blockIdx.x * blockDim.x + tx;
		int column = tid + k + LU_BlockDim;

		if (column < M)
		{
			// Calculate values and write to shared memory array
			for (int r = 0; r < LU_BlockDim; r++) // For each row in block
			{
				float res = a[(r+k) * M + column];
				for (int c = 0; c < r; c++) // 0<=c<r, so below diagonal
					res -= a[(r+k) * M + c + k] * y[tx * LU_BlockDim + c];
				y[tx * LU_BlockDim + r] = res;
			}

			// Copy values from shared memory to global memory
			for (int r = 0; r < LU_BlockDim; r++)
				a[(r+k) * M + column] = y[tx * LU_BlockDim + r];
		}
	}
	// Mat.mult v1, Several blocks with LU_BlockDim x LU_BlockDim threads
	__global__ void lud_block_matrixMultiplication(float *a, int M, int k, int maxDimThreads) {
   
		// Declare cache
		extern __shared__ float shared[];
        
		float* ac = (float*)shared; // Tiling dim should be equal to blockDim.x and blockDim.y
		float* bc = (float*)&shared[blockDim.x*blockDim.y]; // Tiling dim should be equal to blockDim.x and blockDim.y
    
		// Coordinates + row/column end
		const int txid = blockIdx.x * blockDim.x + threadIdx.x; // Column
		const int tyid = blockIdx.y * blockDim.y + threadIdx.y; // Row
    
		float av = 0, bv = 0;

		// Data prefetching + calculate the first index in of row in a.
		if (tyid < maxDimThreads) av = a[(tyid+k+blockDim.y) * M + threadIdx.x + k];

		// Insert register value to shared
		ac[threadIdx.y * blockDim.y + threadIdx.x] = av;

		// Data prefetch + calculate the first index in of column in b.
		if (txid < maxDimThreads) bv = a[(k+threadIdx.y) * M + k + blockDim.x + txid];

		// Calculate index in c, latency hiding
		const int cidx = (tyid+k+blockDim.y) * M + txid+k+blockDim.x;
    
		// Insert register value to shared
		bc[threadIdx.y * blockDim.y + threadIdx.x] = bv;

		// Synchronze to make sure all threads in block have saved values to the shared memory for this phase
		__syncthreads();

		if (txid < maxDimThreads && tyid < maxDimThreads)
		{    
			float sum = 0.0;

			// Calculate the dot-product
			for (int i=0; i < blockDim.x; ++i) {
				sum += ac[threadIdx.y * blockDim.y + i]*bc[i * blockDim.y + threadIdx.x];
			}
        
			// Synchronise to make sure that computation are done
			__syncthreads();
        
			// Insert dot-product in resulting matrix
			a[cidx] -= sum;
		}
	}

	/* Pivot optimised for O(log(N)). Needs 2 kernels. First for reduction on matrix A, 
   sequential reductions are performed on array. TODO: Optimise */
	__global__ void lud_block_pivot(int *out, float *a, int M, int k, int max)
{
    extern __shared__ float shared[];
	float* max_cache = (float*)shared;
	int* idx_cache = (int*)&shared[blockDim.x];

    unsigned int tx = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + tx + k; // Get row index
    
	unsigned int idx = i * M;// + k;

	// Clear cache for threads that exceeds max + they should not influence result
	max_cache[tx] = 0;
	idx_cache[tx] = -1;

	if (i < M)
	{
		// Read value + set row index
		max_cache[tx] = abs(a[idx + k]);
		idx_cache[tx] = i;

		// Sync threads to make sure all other also have loaded values
		__syncthreads();

		// Do the actual pivot finding
		for(unsigned int stride = blockDim.x/2; stride>0; stride>>=1)
		{
			if (tx < stride && (stride+tx+k) < M && max_cache[tx] < max_cache[tx + stride])
			{
				max_cache[tx] = max_cache[tx + stride]; // Update value
				idx_cache[tx] = idx_cache[tx + stride]; // Update index
			}

			// Sync threads
			__syncthreads();
		}

		// The first thread should write result from block to output
		if (tx == 0)
		{
			//out[blockIdx.x][0] = max_cache[0][0]; // Load value to output
			out[blockIdx.x] = idx_cache[0]; // Load index to output
		}
	}
}
	__global__ void lud_block_pivot_L2(int *val, float *a, int M, int k, int max)
	{
		extern __shared__ float shared[];
		float* max_cache = (float*)shared;
		int* idx_cache = (int*)&shared[blockDim.x];

		unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

		unsigned int tx = threadIdx.x;
		unsigned int i = blockIdx.x * blockDim.x + tx; // Get row index
    
		//unsigned int idx = i * M + k;

		// Clear cache for threads that exceeds max + they should not influence result
		max_cache[tx] = 0;

		if (tid < max)
		{
			// Read value + set row index
			int v = val[i];
			max_cache[tx] = abs(a[v * M + k]);
			idx_cache[tx] = v;

			// Sync threads to make sure all other also have loaded values
			__syncthreads();

			// Do the actual pivot finding
			for(unsigned int stride = blockDim.x/2; stride>0; stride>>=1)
			{
				if (tx < stride && max_cache[tx] < max_cache[tx + stride])
				{
					max_cache[tx] = max_cache[tx + stride]; // Update value
					idx_cache[tx] = idx_cache[tx + stride]; // Update index
				}

				// Sync threads
				__syncthreads();
			}

			// The first thread should write result from block to output
			if (tx == 0)
			{
				//out[blockIdx.x][0] = max_cache[0]; // Load value to output
				val[blockIdx.x] = idx_cache[0]; // Load index to output
			}
		}
	}
	// Swap v2, several blocks with 64 (LUDBLOCK_SWAP_BLOCKSIZE2) threads
	__global__ void lud_block_swap_v2(float *a, int M, int r1, int r2)
	{
		int column = blockIdx.x * blockDim.x + threadIdx.x;

		if (column < M)
		{
			float tmp = a[r1 * M + column];
			a[r1 * M + column] = a[r2 * M + column];
			a[r2 * M + column] = tmp;
		}
	}


}

